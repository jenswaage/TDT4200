#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <math.h>

#include "../inc/argument_utils.h"

typedef int64_t int_t;
typedef double real_t;

int_t
    N,
    max_iteration,
    snapshot_frequency;

const real_t
    domain_size = 10.0,
    gravity = 9.81,
    density = 997.0;

// TODO #1.1: Declare device pointers
real_t
    *h_mass[2] = {NULL, NULL},
    *h_mass_velocity_x[2] = {NULL, NULL},
    *h_velocity_x = NULL,
    *h_acceleration_x = NULL,
    *d_mass_0 = {NULL},
    *d_mass_1 = {NULL},
    *d_mass_velocity_x_0 = {NULL},
    *d_mass_velocity_x_1 = {NULL},
    *d_velocity_x = {NULL},
    *d_acceleration_x = {NULL},
    dx,
    dt;

int_t max_threads = 1024;

#define PN(x) mass_0[(x)]
#define PN_next(x) mass_1[(x)]
#define PNU(x) mass_velocity_x_0[(x)]
#define PNU_next(x) mass_velocity_x_1[(x)]
#define U(x) velocity_x[(x)]
#define DU(x) acceleration_x[(x)]

#define cudaErrorCheck(ans)                   \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

// add mass_velocity_x as parameter used in boundary condition
__global__ void time_step_1(real_t *acceleration_x, real_t *mass_0, real_t *mass_velocity_x, real_t *velocity_x, int_t N);
__global__ void time_step_2(real_t *mass_velocity_x_0, real_t *mass_velocity_x_1, real_t *acceleration_x, real_t *mass_0, real_t *mass_1, real_t *velocity_x, real_t dx, real_t dt, int_t N);
__device__ void boundary_condition(real_t *domain_variable, int sign); // only callable from device
void domain_init(void);
void domain_save(int_t iteration);
void domain_finalize(void);

void swap(real_t **m1, real_t **m2)
{
    real_t *tmp;
    tmp = *m1;
    *m1 = *m2;
    *m2 = tmp;
}

int main(int argc, char **argv)
{
    OPTIONS *options = parse_args(argc, argv);
    if (!options)
    {
        fprintf(stderr, "Argument parsing failed\n");
        exit(1);
    }

    N = options->N;
    max_iteration = options->max_iteration;
    snapshot_frequency = options->snapshot_frequency;

    domain_init();

    // TODO #2.4: Define the thread block size and calculate the corresponding grid size.

    int_t block_size = max_threads; // we want the max number of threads in each block

    int_t grid_size = ceil((float_t)(N + 2) / block_size); // we have to account for border indexes

    for (int_t iteration = 0; iteration <= max_iteration; iteration++)
    {   

        // TODO #1.6: Change the function calls to invoke the kernels with the arguments to the device-side equivalents
        time_step_1<<<grid_size, block_size>>>(d_acceleration_x, d_mass_0, d_mass_velocity_x_0, d_velocity_x, N);
        time_step_2<<<grid_size, block_size>>>(d_mass_velocity_x_0, d_mass_velocity_x_1, d_acceleration_x, d_mass_0, d_mass_1, d_velocity_x, dx, dt, N);

        if (iteration % snapshot_frequency == 0)
        {
            printf(
                "Iteration %ld of %ld (%.2lf%% complete)\n",
                iteration,
                max_iteration,
                100.0 * (real_t)iteration / (real_t)max_iteration);

            // TODO #1.8: Copy the results back from the GPU into the host-side buffers.
            hipMemcpy(h_mass[0], d_mass_0, (N + 2) * sizeof(real_t), hipMemcpyDeviceToHost);
            domain_save(iteration);
        }

        // TODO #1.7: Swap device pointers
        swap(&d_mass_0, &d_mass_1);
        swap(&d_mass_velocity_x_0, &d_mass_velocity_x_1);
    }

    domain_finalize();

    exit(EXIT_SUCCESS);
}

// TODO #1.5: Change the host-side function to be a device-side function
__device__ void boundary_condition(real_t *domain_variable, int sign, int_t N, int_t idx) 
{
    #define VAR(x) domain_variable[(x)]

        // we have the first thread set both boundary conditions
        if (idx == 0)
        {
            VAR(0) = sign * VAR(2);
            VAR(N + 1) = sign * VAR(N - 1);
        }

    #undef VAR
}

// TODO #1.4: Change the function to be a CUDA kernel
__global__ void time_step_1(real_t *acceleration_x, real_t *mass_0, real_t *mass_velocity_x, real_t *velocity_x, int_t N)
{

    // TODO #2.1: Define the global index
    int_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x; 

    if (thread_idx > N + 1) {
        return; // this thread does not need to calculate
    }

    // TODO #2.3: Restrict the boundary_condition updates to only be performed by the first and last thread
    // TODO #1.5: Move the boundary_condition calls to inside the time_step_1 kernel
    boundary_condition(mass_0, 1, N, thread_idx); 
    boundary_condition(mass_velocity_x, -1, N, thread_idx);

    DU(thread_idx) = PN(thread_idx) * U(thread_idx) * U(thread_idx) + 0.5 * gravity * PN(thread_idx) * PN(thread_idx) / density;


}

// TODO #1.4: Change the function to be a CUDA kernel
__global__ void time_step_2(real_t *mass_velocity_x_0, real_t *mass_velocity_x_1, real_t *acceleration_x, real_t *mass_0, real_t *mass_1, real_t *velocity_x, real_t dx, real_t dt, int_t N)
{
    // TODO #2.1: Define the global index
    int_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_idx > N || thread_idx < 1) {
        return; // this thread does not need to calculate
    }
    
    boundary_condition(mass_velocity_x_0, -1, N, thread_idx);


    PNU_next(thread_idx) = 0.5 * (PNU(thread_idx + 1) + PNU(thread_idx - 1)) - dt * ((DU(thread_idx + 1) - DU(thread_idx - 1)) / (2 * dx));

    PN_next(thread_idx) = 0.5 * (PN(thread_idx + 1) + PN(thread_idx - 1)) - dt * ((PNU(thread_idx + 1) - PNU(thread_idx - 1)) / (2 * dx));

    U(thread_idx) = PNU_next(thread_idx) / PN_next(thread_idx);
    
}

void domain_init(void)
{
    // TODO #1.2: Allocate space for the device-side buffers.

    h_mass[0] = (real_t *)calloc((N + 2), sizeof(real_t));
    h_mass[1] = (real_t *)calloc((N + 2), sizeof(real_t));

    hipMalloc((void **)&d_mass_0, (N + 2) * sizeof(real_t));
    hipMalloc((void **)&d_mass_1, (N + 2) * sizeof(real_t));

    h_mass_velocity_x[0] = (real_t *)calloc((N + 2), sizeof(real_t));
    h_mass_velocity_x[1] = (real_t *)calloc((N + 2), sizeof(real_t));

    hipMalloc((void **)&d_mass_velocity_x_0, (N + 2) * sizeof(real_t));
    hipMalloc((void **)&d_mass_velocity_x_1, (N + 2) * sizeof(real_t));

    h_velocity_x = (real_t *)calloc((N + 2), sizeof(real_t));

    hipMalloc((void **)&d_velocity_x, (N + 2) * sizeof(real_t));

    h_acceleration_x = (real_t *)calloc((N + 2), sizeof(real_t));

    hipMalloc((void **)&d_acceleration_x, (N + 2) * sizeof(real_t));

    // Data initialization
    for (int_t x = 1; x <= N; x++)
    {
        h_mass[0][(x)] = 1e-3;
        h_mass_velocity_x[0][(x)] = 0.0;

        real_t c = x - (double)N / 2;
        if (sqrt(c * c) < N / 20.0)
        {
            h_mass[0][(x)] -= 5e-4 * exp(-4 * pow(c, 2.0) / (real_t)(N));
        }

        h_mass[0][(x)] *= density;
    }

    dx = domain_size / (real_t)N;
    dt = 0.1 * dx;

    // TODO #1.3: Transfer the host-side data to the device.
    hipMemcpy(d_mass_0, h_mass[0], (N + 2) * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_1, h_mass[1], (N + 2) * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_velocity_x_0, h_mass_velocity_x[0], (N + 2) * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_velocity_x_1, h_mass_velocity_x[1], (N + 2) * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_velocity_x, h_velocity_x, (N + 2) * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_acceleration_x, h_acceleration_x, (N + 2) * sizeof(real_t), hipMemcpyHostToDevice);
}

void domain_save(int_t iteration)
{
    int_t index = iteration / snapshot_frequency;
    char filename[256];
    memset(filename, 0, 256 * sizeof(char));
    sprintf(filename, "data/%.5ld.bin", index);

    FILE *out = fopen(filename, "wb");
    if (!out)
    {
        fprintf(stderr, "Failed to open file: %s\n", filename);
        exit(1);
    }
    fwrite(&h_mass[0][1], sizeof(real_t), N, out);
    fclose(out);
}

void domain_finalize(void)
{
    free(h_mass[0]);
    free(h_mass[1]);
    free(h_mass_velocity_x[0]);
    free(h_mass_velocity_x[1]);
    free(h_velocity_x);
    free(h_acceleration_x);

    // TODO #1.9: Add hipFree for the device buffers.
    hipFree(d_mass_0);
    hipFree(d_mass_1);
    hipFree(d_mass_velocity_x_0);
    hipFree(d_mass_velocity_x_1);
    hipFree(d_velocity_x);
    hipFree(d_acceleration_x);
}
